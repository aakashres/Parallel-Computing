/*
 *  file name: matrix.cu
 *
 *  matrix.cu contains the code that realize some common used matrix operations in CUDA
 *
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < n && row < n)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int n)
{
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * n + j];
            }
            h_result[i * n + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int n = 10;
    int *h_a, *h_b, *h_c, *h_cc;
    int *d_a, *d_b, *d_c;
    
    /* Fixed seed for illustration */
    srand(3333);
    // allocate memory in host RAM, h_cc is used to store CPU result
    hipHostMalloc((void **)&h_a, sizeof(int) * n * n, hipHostMallocDefault);
    hipHostMalloc((void **)&h_b, sizeof(int) * n * n, hipHostMallocDefault);
    hipHostMalloc((void **)&h_c, sizeof(int) * n * n, hipHostMallocDefault);
    hipHostMalloc((void **)&h_cc, sizeof(int) * n * n, hipHostMallocDefault);

    // random initialize matrix A
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_b[i * n + j] = rand() % 1024;
        }
    }

    // Allocate memory space on the device
    hipMalloc((void **)&d_a, sizeof(int) * n * n);
    hipMalloc((void **)&d_b, sizeof(int) * n * n);
    hipMalloc((void **)&d_c, sizeof(int) * n * n);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * n, hipMemcpyHostToDevice);

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    // Transefr results from device to host
    hipMemcpy(h_c, d_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);
    cpu_matrix_mult(h_a, h_b, h_cc, n);
    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            // printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_cc[i*k + j], i, j, h_c[i*k + j]);
            if (h_cc[i * n + j] != h_c[i * n + j])
            {
                all_ok = 0;
            }
        }
        // printf("\n");
    }

    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", h_c[i*n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", h_cc[i*n + j]);
        }
        printf("\n");
    }


    // roughly compute speedup
    if (all_ok)
    {
        printf("all results are correct!!!");
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}